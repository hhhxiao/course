#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"

#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;
void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__ void bodyForce(Body *p, float dt, int n) {
  //遍历每个物品
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < n; i += stride)
  {
  float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;
      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }
    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }  
}

int main(const int argc, const char** argv) {
  int nBodies = 2<<11;
  int salt = 0;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);
  if (argc > 2) salt = atoi(argv[2]);
  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations
  //获取GPU信息
  int deviceId;
  int numberOfSMs;
  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  int bytes = nBodies * sizeof(Body);
  float *buf;
  hipMallocManaged(&buf, bytes);
  Body *p = (Body*)buf;
  hipMemPrefetchAsync(buf, bytes, hipCpuDeviceId);
  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data
  double totalTime = 0.0;
  // Do not modify these 2 lines of code.
  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();
  /*******************************************************************/

  /*
   * You will likely wish to refactor the work being done in `bodyForce`,
   * as well as the work to integrate the positions.
   */
    hipMemPrefetchAsync(buf, bytes, deviceId);
    bodyForce<<<32*numberOfSMs,64>>>(p, dt, nBodies); // compute interbody forces
    
      
      hipDeviceSynchronize();
    //更新坐标
    hipMemPrefetchAsync(buf, bytes, hipCpuDeviceId);
    for (int i = 0 ; i < nBodies; i++) { // integrate position
    p[i].x += p[i].vx*dt;
    p[i].y += p[i].vy*dt;
    p[i].z += p[i].vz*dt;
    }
    //下面是计算平均时间
  /*******************************************************************/
  // Do not modify the code in this section.
    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }
  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
  checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
  checkAccuracy(buf, nBodies);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
  salt += 1;
#endif
  /*******************************************************************/

  /*
   * Feel free to modify code below.
   */
  hipFree(buf);
 // free(buf);
}
